#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include "Snapshot.h"
#include "Agent.h"

//those static values are CPU and GPU counterpart variables. usually in GPU variable start with dev_(device)
static bool *Gdir=NULL,*dev_dir;//dir is DIR in python
static double *Gweights,*dev_weights,*Gthresholds,*dev_thresholds;//weight and threshold in python
static bool *Gobserve,*dev_observe;//observe in python
static bool *Gdfs,*dev_dfs;//this variable is a bool value used in dfs function
static bool *Gsignal,*dev_signal,*Gload,*dev_load;//signal and load variable in propagate
static bool *Gcurrent,*dev_current;//current in python
static bool *Gmask,*dev_mask;//bool value for mask signal in halucinate
static int *tmp_signal,*tmp_load;//tmp variable for dfs on GPU, those two variable are mainly used in bool2int and int2bool, which are tricky ways to mark 'visited' in dfs
static bool *dfs_flag,*dev_dfs_flag;//variable to mark whether new dfs visited occur

//CUDA introduction
/*
in CUDA:
__host__ means the function happens on CPU
__device__ means the function happens on Device(GPU)
__host__ __device__ means the function can be used both on CPU and GPU(the compi on both CPU and GPU)
__global__ means it is an global function. global function is where parallel happens.
	Many threads runs the same code in global function, the way to distinguish different thread is by block and thread
	blockIdx means block ID, threadIdx means thread ID. In CUDA, many threads form a block, many blocks form a grid, you can find the detail in the ppt I sent you
	blockDim is the dimension of block. block and thread can be in multiply dimension(see in ppt)
	to call a global function, you need to specify block number and thread num within each block like:
		fun<<<blockNum,threadNum>>>(para1,para2...), blockNum and threadNum can be in multiply dimension
	when you are accessing the data, make sure they do not go beyond boundary(that is why I have lots of "size" variable to check in almost every global function)
	in global function you can use __device__ function or __host__ __device__ function as long as EVERY VARIABLE IS ON GPU. This is a very strict rule, you cannot access CPU memory on GPU, neither can you access GPU memory on CPU.
hipMalloc(&variable_address,size*sizeof(data_type)):
	The function is used to malloc space for variable on GPU
hipMemcpy(&copy_to_address,&copy_from_address,size*sizeof(data_type),tag):
	The function is copy data from one place to another.
	tag has: hipMemcpyHostToDevice,hipMemcpyDeviceToHost,hipMemcpyDeviceToDevice,hipMemcpyHostToHost
hipMemset(&variable_address,value,size*sizeof(data_type)):
	The function is like memset in C++, give the same value to an address of data
hipFree(&variable_address):
	The function is used to free a variable, like delete in C++

I believe those functions above is enough for the project, if you have any question just email me
*/

//helper function
/*
*/
__host__ __device__ int compi_GPU(int x){
	if(x%2==0) return x+1;
	else return x-1;
}

__host__ __device__ int ind(int row,int col,int width){
	return row*width+col;
}

__global__ void conjunction_kernel(bool *b1,bool *b2,int size){
	int index=blockDim.x*blockIdx.x+threadIdx.x;
	if(index<size){
		b1[index]=b1[index]&&b2[index];
	}
}

__global__ void disjunction_kernel(bool *b1,bool *b2,int size){
	int index=blockDim.x*blockIdx.x+threadIdx.x;
	if(index<size){
		b1[index]=b1[index]||b2[index];
	}
}


__global__ void negate_disjunction_star_kernel(bool *b1,bool *b2,int size){
	int index=blockDim.x*blockIdx.x+threadIdx.x;
	if(index<size){
		if(index%2==0){
			b1[index]=b1[index]&&!b2[index+1];
		}
		else{
			b1[index]=b1[index]&&!b2[index-1];
		}
	}
}

__global__ void int2bool_kernel(bool *b,int *i,int size){
	int index=blockDim.x*blockIdx.x+threadIdx.x;
	if(index<size){
		if(i[index]==1) b[index]=true;
		else b[index]=false;
	}
}

__global__ void bool2int_kernel(int *i,bool *b,int size){
	int index=blockDim.x*blockIdx.x+threadIdx.x;
	if(index<size){
		if(b[index]) i[index]=0;
		else i[index]=-1;
	}
}

//helper function

__device__ bool implies_GPU(int row,int col,int width,double *weights,double threshold){//implies
	double rc=weights[ind(row,col,width)];
	double r_c=weights[ind(compi_GPU(row),col,width)];
	double rc_=weights[ind(row,compi_GPU(col),width)];
	double r_c_=weights[ind(compi_GPU(row),compi_GPU(col),width)];
	double epsilon=(rc+r_c+rc_+r_c_)*threshold;
	double m=min(epsilon,min(rc,min(r_c,r_c_)));
	return rc_<m;
}

__device__ bool equivalent_GPU(int row,int col,int width,double *weights,double threshold){//equivalent
	double rc=weights[ind(row,col,width)];
	double r_c=weights[ind(compi_GPU(row),col,width)];
	double rc_=weights[ind(row,compi_GPU(col),width)];
	double r_c_=weights[ind(compi_GPU(row),compi_GPU(col),width)];
	double epsilon=(rc+r_c+rc_+r_c_)*threshold;
	return rc_==0&&r_c==0;
}

__device__ void orient_square_GPU(bool *dir,double *weights,double *thresholds,int x,int y,int width){//orient_square
	dir[ind(x,y,width)]=false;
	dir[ind(x,compi_GPU(y),width)]=false;
	dir[ind(compi_GPU(x),y,width)]=false;
	dir[ind(compi_GPU(x),compi_GPU(y),width)]=false;
	dir[ind(y,x,width)]=false;
	dir[ind(compi_GPU(y),x,width)]=false;
	dir[ind(y,compi_GPU(x),width)]=false;
	dir[ind(compi_GPU(y),compi_GPU(x),width)]=false;

	int square_is_oriented=0;
	for(int i=0;i<2;++i){
		for(int j=0;j<2;++j){
			int sx=x+i;
            int sy=y+j;
			if(square_is_oriented==0){
				if(implies_GPU(sy,sx,width,weights,thresholds[ind(sy,sx,width)])){
					dir[ind(sy,sx,width)]=true;
					dir[ind(compi_GPU(sx),compi_GPU(sy),width)]=true;
					dir[ind(sx,sy,width)]=false;
                    dir[ind(compi_GPU(sy),compi_GPU(sx),width)]=false;
                    dir[ind(sx,compi_GPU(sy),width)]=false;
                    dir[ind(compi_GPU(sy),sx,width)]=false;
                    dir[ind(sy,compi_GPU(sx),width)]=false;
                    dir[ind(compi_GPU(sx),sy,width)]=false;
                    square_is_oriented=1;
				}//implies
				if(equivalent_GPU(sy,sx,width,weights,thresholds[ind(sy,sx,width)])){
					dir[ind(sy,sx,width)]=true;
					dir[ind(sx,sy,width)]=true;
					dir[ind(compi_GPU(sx),compi_GPU(sy),width)]=true;
                    dir[ind(compi_GPU(sy),compi_GPU(sx),width)]=true;
					dir[ind(sx,compi_GPU(sy),width)]=false;
                    dir[ind(compi_GPU(sy),sx,width)]=false;
                    dir[ind(sy,compi_GPU(sx),width)]=false;
                    dir[ind(compi_GPU(sx),sy,width)]=false;
                    square_is_oriented=1;
				}//equivalent
			}//square_is_oriented
		}//j
	}//i
}

__global__ void update_weights_kernel(double *weights,bool *observe,int size){
	int indexX=blockDim.x*blockIdx.x+threadIdx.x;
	int indexY=blockDim.y*blockIdx.y+threadIdx.y;
	if(indexX<size&&indexY<size){
		weights[ind(indexY,indexX,size)]+=observe[indexX]*observe[indexY];
	}
}

__global__ void orient_all_kernel(bool *dir,double *weights,double *thresholds,int size){
	int indexX=blockDim.x*blockIdx.x+threadIdx.x;
	int indexY=blockDim.y*blockIdx.y+threadIdx.y;
	//the commented code is the optimization for the triangle problem we discussed. I think the speed is fast for now so I just use the original one
	/*if(indexX<size){//possible optimazation
		if(indexY>indexX) orient_square_GPU(dir,weights,thresholds,2*(size/2-1-indexX),2*(size/2-1-indexY),size);
		else if(indexY<indexX) orient_square_GPU(dir,weights,thresholds,2*indexX,2*indexY,size);
	}*/
	if(indexX<size/2&&indexY<indexX){
		orient_square_GPU(dir,weights,thresholds,indexX*2,indexY*2,size);
	}
}

__global__ void dfs_GPU(bool *dir,int *dfs,bool *flag,int size){//dfs
	int index=blockDim.x*blockIdx.x+threadIdx.x;
	if(index<size&&dfs[index]==0){
		for(int j=0;j<size;++j){
			if(j==index) continue;
			if(dir[ind(index,j,size)]){
				atomicMax(dfs+j,0);
				flag[0]=true;
			}
		}
		//the atomic function in CUDA make sure only one operation is done at one time, like a lock in C++ or Java.
		//http://docs.nvidia.com/cuda/cuda-c-programming-guide/index.html#atomic-functions
		atomicMax(dfs+index,1);
	}
}

//mask=Signal([(ind in actions_list) for ind in xrange(self._SIZE)])
__global__ void mask_kernel(bool *mask,int *actionlist,int size){
	int index=blockDim.x*blockIdx.x+threadIdx.x;
	if(index<size){
		for(int i=0;i<size;++i){
			if(index==actionlist[i]){
				mask[index]=true;
				return;
			}
		}
		mask[index]=false;
	}
}

//before invoke this function make sure dev_load and dev_signal have correct data
//the computed data will be in dev_load
void Snapshot::propagate_GPU(){//propagate
	bool2int_kernel<<<(size+255)/256,256>>>(tmp_load,dev_load,size);
	bool2int_kernel<<<(size+255)/256,256>>>(tmp_signal,dev_signal,size);

	dfs_flag[0]=true;
	while(dfs_flag[0]){
		hipMemset(dev_dfs_flag,false,sizeof(bool));
		dfs_GPU<<<(size+255)/256,256>>>(dev_dir,tmp_load,dev_dfs_flag,size);
		hipMemcpy(dfs_flag,dev_dfs_flag,sizeof(bool),hipMemcpyDeviceToHost);
	}
	int2bool_kernel<<<(size+255)/256,256>>>(dev_load,tmp_load,size);
	//load=self.up(load)
	dfs_flag[0]=true;
	while(dfs_flag[0]){
		hipMemset(dev_dfs_flag,false,sizeof(bool));
		dfs_GPU<<<(size+255)/256,256>>>(dev_dir,tmp_signal,dev_dfs_flag,size);
		hipMemcpy(dfs_flag,dev_dfs_flag,sizeof(bool),hipMemcpyDeviceToHost);
	}
	int2bool_kernel<<<(size+255)/256,256>>>(dev_signal,tmp_signal,size);
	//mask_pos=self.up(signal)
	
	disjunction_kernel<<<(size+255)/256,256>>>(dev_load,dev_signal,size);
	negate_disjunction_star_kernel<<<(size+255)/256,256>>>(dev_load,dev_signal,size);
	
	hipMemcpy(Gload,dev_load,size*sizeof(bool),hipMemcpyDeviceToHost);
}

void Snapshot::setSignal(vector<bool> observe){//this is where data comes in in every frame
	for(int i=0;i<observe.size();++i){
		Gobserve[i]=observe[i];
	}
	hipMemcpy(dev_observe,Gobserve,size*sizeof(bool),hipMemcpyHostToDevice);
}

void Snapshot::update_state_GPU(bool mode){//true for decide
	dim3 dimGrid((size+15)/16,(size+15)/16);
	dim3 dimBlock(16,16);
	update_weights_kernel<<<dimGrid,dimBlock>>>(dev_weights,dev_observe,size);
	//update_weight
	
	if(mode){
		dim3 dimGrid1((size/2+15)/16,(size/2+15)/16);
		dim3 dimBlock1(16,16);
		orient_all_kernel<<<dimGrid1,dimBlock1>>>(dev_dir,dev_weights,dev_thresholds,size);
	}//orient_all

	hipMemcpy(dev_signal,dev_observe,size*sizeof(bool),hipMemcpyDeviceToDevice);
	hipMemset(dev_load,false,size*sizeof(bool));
	propagate_GPU();
	hipMemcpy(Gcurrent,dev_load,size*sizeof(bool),hipMemcpyDeviceToHost);
	hipMemcpy(dev_current,dev_load,size*sizeof(bool),hipMemcpyDeviceToDevice);
	hipMemcpy(Gdir,dev_dir,size*size*sizeof(bool),hipMemcpyDeviceToHost);
}

void Snapshot::halucinate_GPU(vector<int> actions_list){
	//mask=Signal([(ind in actions_list) for ind in xrange(self._SIZE)])
	vector<bool> mask=initMask(actions_list);
	vector<int> v;
	for(int i=0;i<actions_list.size();++i){
		for(int j=0;j<size;++j){
			if(context.find(pair<int,int>(actions_list[i],j))!=context.end()&&Gcurrent[j]){
				v.push_back(context[pair<int,int>(actions_list[i],j)]);
			}
		}
	}
	//relevant_pairs=[(act,ind) for act in actions_list for ind in xrange(self._SIZE) if (act,ind) in self._CONTEXT and self._CURRENT.value(ind)]
	//map(mask.set,[self._CONTEXT[i,j] for i,j in relevant_pairs],[True for i,j in relevant_pairs])
	for(int i=0;i<v.size();++i) mask[v[i]]=true;
	
	for(int i=0;i<mask.size();++i){
		Gmask[i]=mask[i];
	}
	hipMemcpy(dev_mask,Gmask,size*sizeof(bool),hipMemcpyHostToDevice);
	//copy data
	hipMemcpy(dev_signal,dev_mask,size*sizeof(bool),hipMemcpyDeviceToDevice);
	hipMemcpy(dev_load,dev_current,size*sizeof(bool),hipMemcpyDeviceToDevice);
	propagate_GPU();
	//return self.propagate(mask,self._CURRENT)
}

void Snapshot::freeData(){//free data in case of memory leak
	delete[] Gdir;
	delete[] Gweights;
	delete[] Gthresholds;
	delete[] Gobserve;
	delete[] Gdfs;
	delete[] Gsignal;
	delete[] Gload;
	delete[] tmp_load;
	delete[] tmp_signal;
	delete[] dfs_flag;
	delete[] Gmask;
	delete[] Gcurrent;
	hipFree(dev_dir);
	hipFree(dev_thresholds);
	hipFree(dev_weights);
	hipFree(dev_observe);
	hipFree(dev_dfs);
	hipFree(dev_signal);
	hipFree(dev_load);
	hipFree(tmp_signal);
	hipFree(tmp_load);
	hipFree(dev_dfs_flag);
	hipFree(dev_mask);
	hipFree(dev_current);
}

void Snapshot::initData(string name,int size,double threshold,vector<vector<int> > context_key,vector<int> context_value,
		vector<string> sensors_names,vector<string> evals_names,vector<vector<int> > generalized_actions){
	//data init
	this->name=name;
	this->size=size;
	this->threshold=threshold;
	this->sensors_names=sensors_names;
	this->evals_names=evals_names;
	this->generalized_actions=generalized_actions;
	srand (time(NULL));
	for(int i=0;i<size;++i){
		name_to_num[sensors_names[i]]=i;
	}
	if(Gdir!=NULL){
		freeData();
	}
	
	Gdir=new bool[size*size];
	Gweights=new double[size*size];
	Gthresholds=new double[size*size];
	Gobserve=new bool[size];
	Gdfs=new bool[1];
	Gsignal=new bool[size];
	Gload=new bool[size];
	tmp_signal=new int[size];
	tmp_load=new int[size];
	dfs_flag=new bool[1];
	Gmask=new bool[size];
	Gcurrent=new bool[size];
	
	hipMalloc(&dev_dir,size*size*sizeof(bool));
	hipMalloc(&dev_thresholds,size*size*sizeof(double));
	hipMalloc(&dev_weights,size*size*sizeof(double));
	hipMalloc(&dev_observe,size*sizeof(bool));
	hipMalloc(&dev_dfs,sizeof(bool));
	hipMalloc(&dev_signal,size*sizeof(bool));
	hipMalloc(&dev_load,size*sizeof(bool));
	hipMalloc(&tmp_signal,size*sizeof(int));
	hipMalloc(&tmp_load,size*sizeof(int));
	hipMalloc(&dev_dfs_flag,sizeof(bool));

	hipMalloc(&dev_mask,size*sizeof(bool));
	hipMalloc(&dev_current,size*sizeof(bool));

	for(int i=0;i<size;++i){
		for(int j=0;j<size;++j){
			Gthresholds[i*size+j]=threshold;
			Gweights[i*size+j]=0.0;
			Gdir[i*size+j]=false;
		}
	}

	hipMemcpy(dev_thresholds,Gthresholds,size*size*sizeof(double),hipMemcpyHostToDevice);
	hipMemcpy(dev_weights,Gweights,size*size*sizeof(double),hipMemcpyHostToDevice);
	hipMemcpy(dev_dir,Gdir,size*size*sizeof(bool),hipMemcpyHostToDevice);
	//init threshold

	for(int i=0;i<context_key.size();++i){
		context[pair<int,int>(context_key[i][0],context_key[i][1])]=context_value[i];
	}
	cout<<"succeed"<<endl;
}

//those three functions down there are get functions for the variable in C++
vector<bool> Snapshot::getCurrent(){
	vector<bool> result;
	for(int i=0;i<size;++i){
		result.push_back(Gcurrent[i]);
	}
	return result;
}

vector<bool> Snapshot::getLoad(){
	vector<bool> result;
	for(int i=0;i<size;++i){
		result.push_back(Gload[i]);
	}
	return result;
}

vector<vector<bool> > Snapshot::getDir(){
	vector<vector<bool> > result;
	for(int i=0;i<size;++i){
		vector<bool> tmp;
		for(int j=0;j<size;++j){
			tmp.push_back(Gdir[i*size+j]);
		}
		result.push_back(tmp);
	}
	return result;
}